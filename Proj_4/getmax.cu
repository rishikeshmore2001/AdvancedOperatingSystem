#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void reduce_max_kernel(double *g_idata, double *g_odata, int n) {
    extern __shared__ double sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sdata[tid] = (i < n) ? g_idata[i] : -1e9;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (sdata[tid + s] > sdata[tid]) {
                sdata[tid] = sdata[tid + s];
            }
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

// Internal: perform max reduction on GPU
double find_max_gpu(double *h_arr, int size) {
    double *d_in, *d_out;
    double max_val = -1e9;
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    int out_size = blocks;

    // Allocate device memory
    hipMalloc(&d_in, size * sizeof(double));
    hipMalloc(&d_out, blocks * sizeof(double));

    // Copy data to device
    hipMemcpy(d_in, h_arr, size * sizeof(double), hipMemcpyHostToDevice);

    // First reduction pass
    reduce_max_kernel<<<blocks, threads, threads * sizeof(double)>>>(d_in, d_out, size);
    hipDeviceSynchronize();

    // Continue reducing until one block remains
    while (out_size > 1) {
        int new_blocks = (out_size + threads - 1) / threads;
        reduce_max_kernel<<<new_blocks, threads, threads * sizeof(double)>>>(d_out, d_out, out_size);
        hipDeviceSynchronize();
        out_size = new_blocks;
    }

    // Copy final result back to host
    hipMemcpy(&max_val, d_out, sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_in);
    hipFree(d_out);

    return max_val;
}

// RPC-callable: generate data + return max from GPU
extern "C"
double getmax_cpu(int N, double M, int seed) {
    int size = 1 << N;  // 2^N
    double *arr = (double *)malloc(size * sizeof(double));

    if (!arr) {
        fprintf(stderr, "Host allocation failed\n");
        return -1.0;
    }

    // Initialize array with exponential distribution
    srand(seed);
    for (int i = 0; i < size; ++i) {
        double U = (double)rand() / RAND_MAX;
        arr[i] = -M * log(1.0 - U);  // Exponential distribution
    }

    // Launch GPU kernel to compute max
    double result = find_max_gpu(arr, size);

    free(arr);
    return result;
}
